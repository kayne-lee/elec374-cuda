/* Kayne Lee, 20350003 */

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include <time.h>

#define NUM_SIZES 5

// different matrix sizes to test
int sizes[NUM_SIZES] = { 256, 512, 1024, 2048, 4096 };

// function to measure memory transfer time (part 1 of part 2)
void measureTransferTime(int n) {
    size_t bytes = n * n * sizeof(float);

    // allocate host memory
    float* h_matrix = (float*)malloc(bytes);
    for (int i = 0; i < n * n; i++) {
        h_matrix[i] = (float)(rand() % 100);
    }

    // allocate device memory
    float* d_matrix;
    hipMalloc((void**)&d_matrix, bytes);

    // create CUDA events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float timeH2D = 0.0f, timeD2H = 0.0f;

    // measure Host to Device (H2D) transfer time
    hipEventRecord(start, 0);
    hipMemcpyAsync(d_matrix, h_matrix, bytes, hipMemcpyHostToDevice, 0);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&timeH2D, start, stop);

    // measure Device to Host (D2H) transfer time
    hipEventRecord(start, 0);
    hipMemcpyAsync(h_matrix, d_matrix, bytes, hipMemcpyDeviceToHost, 0);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&timeD2H, start, stop);

    // print results
    printf("Matrix Size: %d x %d | H2D Transfer: %.3f ms | D2H Transfer: %.3f ms\n", n, n, timeH2D, timeD2H);

    // cleanup
    free(h_matrix);
    hipFree(d_matrix);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

// Function to multiply matrices on the CPU
void multiplyMatricesCPU(float* A, float* B, float* C, int n) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            C[i * n + j] = 0;
            for (int k = 0; k < n; k++) {
                C[i * n + j] += A[i * n + k] * B[k * n + j];
            }
        }
    }
}

// CUDA kernel for matrix multiplication (single block, one thread)
__global__ void multiplyMatricesGPU(float* A, float* B, float* C, int n) {
    int i = 0, j = 0;
    for (i = 0; i < n; i++) {
        for (j = 0; j < n; j++) {
            float sum = 0.0f;
            for (int k = 0; k < n; k++) {
                sum += A[i * n + k] * B[k * n + j];
            }
            C[i * n + j] = sum;
        }
    }
}

// Function to measure CPU matrix multiplication time
void measureMatrixMultiplicationCPU(int n) {
    size_t bytes = n * n * sizeof(float);
    float* h_A = (float*)malloc(bytes);
    float* h_B = (float*)malloc(bytes);
    float* h_C = (float*)malloc(bytes);

    for (int i = 0; i < n * n; i++) {
        h_A[i] = (float)(rand() % 100);
        h_B[i] = (float)(rand() % 100);
    }

    clock_t start = clock();
    multiplyMatricesCPU(h_A, h_B, h_C, n);
    clock_t end = clock();

    printf("CPU Matrix Multiplication Time for %d x %d: %.6f seconds\n", n, n, (double)(end - start) / CLOCKS_PER_SEC);

    free(h_A);
    free(h_B);
    free(h_C);
}

// Function to measure GPU matrix multiplication time (including transfer time)
void measureMatrixMultiplicationGPU(int n) {
    size_t bytes = n * n * sizeof(float);
    float* h_A = (float*)malloc(bytes);
    float* h_B = (float*)malloc(bytes);
    float* h_C = (float*)malloc(bytes);
    float* d_A, * d_B, * d_C;

    for (int i = 0; i < n * n; i++) {
        h_A[i] = (float)(rand() % 100);
        h_B[i] = (float)(rand() % 100);
    }

    hipMalloc((void**)&d_A, bytes);
    hipMalloc((void**)&d_B, bytes);
    hipMalloc((void**)&d_C, bytes);

    // Measure H2D transfer time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float timeH2D = 0.0f;
    hipEventRecord(start, 0);
    hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&timeH2D, start, stop);

    // Launch the GPU kernel
    hipEventRecord(start, 0);
    multiplyMatricesGPU <<<1, 1 >>> (d_A, d_B, d_C, n);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float kernelTime;
    hipEventElapsedTime(&kernelTime, start, stop);

    // Measure D2H transfer time
    float timeD2H = 0.0f;
    hipEventRecord(start, 0);
    hipMemcpy(h_C, d_C, bytes, hipMemcpyDeviceToHost);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&timeD2H, start, stop);

    printf("GPU Matrix Multiplication Time for %d x %d: %.6f seconds (kernel: %.6f + H2D: %.3f + D2H: %.3f)\n",
        n, n, (kernelTime + timeH2D + timeD2H) / 1000.0f, kernelTime / 1000.0f, timeH2D, timeD2H);

    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

int main() {
    srand(time(NULL));

    printf("Measuring Memory Transfer Time (Host ↔ Device)\n");
    printf("---------------------------------------------------\n");

    for (int i = 0; i < NUM_SIZES; i++) {
        measureTransferTime(sizes[i]);
    }

    printf("\nMatrix Multiplication CPU vs GPU\n");
    printf("---------------------------------\n");
    for (int i = 0; i < NUM_SIZES; i++) {
        int size = sizes[i];
        measureMatrixMultiplicationCPU(size);
        measureMatrixMultiplicationGPU(size);
    }

    return 0;
}
